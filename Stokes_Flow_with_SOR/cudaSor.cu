#include "hip/hip_runtime.h"
//Lionel Fiske
//Cuda SOR




#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

//void KernelU(double* dev_U, double* dev_maxResU, int* dev_N, double* dev_dx, double* dev_mu);


static void HandleError(hipError_t err, const char* file, int line){

	if(err != hipSuccess){
		printf("%s in %s at line %d \n ", hipGetErrorString(err), file, line );
		exit(1);
	}
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__,__LINE__))


//__global__ void KernelU(double* dev_U,double* dev_P, double* dev_ResU,double* dev_numberOfBlocksU , int* dev_N, double* dev_P_const, double* dev_dx, double* dev_mu, double* dev_omega){

//__shared__ double localU[pointsPerBlock];

/*
//Since we are not interested in the residual I will keep resU resV and resP locally on card
double uC=100;
double uE=100;
double uW=100;
double uN=100;
double uS=100;
double pC=100;
double pW=100;
double P_const=dev_P_const[0];
double omega=*dev_omega;

int i;
int j;
int gi;
int gj;
int rowsPerBlock=dev_N[0]/ dev_numberOfBlocksU[0]; //The local block cN
int dx=dev_dx[0];
int mu=dev_mu[0];
int N=dev_N[0];
int g_tid=(blockIdx.x)*blockDim.x+threadIdx.x;
int tid=threadIdx.x;



for(int color=1; color<3; color++){

//Compute U Residual and update U

//Not in first row or last row 	

	if(blockIdx.x!=0 && blockIdx.x!=*dev_numberOfBlocksU){	
		if(N/2<tid<N/2*(rowsPerBlock) ){

			gi=tid%dev_N; 
			gj=tid/N;

			i=tid%N; //U has N cols
			j=tid/N;

			i=2*i + (j+color)%2; //Color selector 

			//Center points of stencil 
			uC=dev_U[i+N*j];
			pC=dev_P[i+(N-1)*j]; 


			//Handle the x left boundary 
			if(gi!=0 ){
				uW=dev_U[i-1+N*j];
				pW=dev_P[(i-1)+(N-1)*j];
			}

			//left points
			if(gi==0 ){
				uW=uC;
				pW=2*P_const - pC;
			}

			//handle x right interior points
			if(gi!=N-1){
			uE=dev_U[i+1+N*j];
			}

			//handle x right boundary points
			if(gi==N-1){
			uE=uC;
			pC=-dev_P[i-1+(N-1)*j]; 
			}


			//Handle the up interior point
			if(gj!=0 ){
				uN=dev_U[i+N*(j-1)];
			}

			//Handle the y top boundary case
			if(gj==0 ){
				uN=-uC;
			}

			//Handle the y down interior case
			if(gj!=(N-1)-1 ){
				uS=dev_U[i+N*(j+1)];
			}

			//Handle the y down boundary case
			if(gj==(N-1)-1 ){
				uS=-uC;
			}

			// Compute Residual 
				resU= mu*(uE + uW - 2*uC)+ mu*(uN + uS - 2*uC) - dx*(pC-pW);
				dev_U[i+N*j]=dev_U[i+N*j]+ omega*resU;


			}

		}



	if(blockIdx.x==0){	
		if(N/2<tid<N/2*(rowsPerBlock) ){

			gi=tid%dev_N; 
			gj=tid/N;

			i=tid%N; //U has N cols
			j=tid/N;

			i=2*i + (j+color)%2; //Color selector 

			//Center points of stencil 
			uC=dev_U[i+N*j];
			pC=dev_P[i+(N-1)*j]; 


			//Handle the x left boundary 
			if(gi!=0 ){
				uW=dev_U[i-1+N*j];
				pW=dev_P[(i-1)+(N-1)*j];
			}

			//left points
			if(gi==0 ){
				uW=uC;
				pW=2*P_const - pC;
			}

			//handle x right interior points
			if(gi!=N-1){
			uE=dev_U[i+1+N*j];
			}

			//handle x right boundary points
			if(gi==N-1){
			uE=uC;
			pC=-dev_P[i-1+(N-1)*j]; 
			}


			//Handle the up interior point
			if(gj!=0 ){
				uN=dev_U[i+N*(j-1)];
			}

			//Handle the y top boundary case
			if(gj==0 ){
				uN=-uC;
			}

			//Handle the y down interior case
			if(gj!=(N-1)-1 ){
				uS=dev_U[i+N_col*(j+1)];
			}

			//Handle the y down boundary case
			if(gj==(N-1)-1 ){
				uS=-uC;
			}

			// Compute Residual 
				resU= mu*(uE + uW - 2*uC)+ mu*(uN + uS - 2*uC) - dx*(pC-pW);
				dev_U[i+N*j]=dev_U[i+N*j]+ omega*resU;

			//Check if this is the biggest residual 


			}

		}


	if(blockIdx.x==*dev_numberOfBlocksU){	
		if(N/2<tid<N/2*(rowsPerBlock) ){

			gi=tid%dev_N; 
			gj=tid/N;

			i=tid%N; //U has N cols
			j=tid/N;

			i=2*i + (j+color)%2; //Color selector 

			//Center points of stencil 
			uC=dev_U[i+N*j];
			pC=dev_P[i+(N-1)*j]; 


			//Handle the x left boundary 
			if(gi!=0 ){
				uW=dev_U[i-1+N*j];
				pW=dev_P[(i-1)+(N-1)*j];
			}

			//left points
			if(gi==0 ){
				uW=uC;
				pW=2*P_const - pC;
			}

			//handle x right interior points
			if(gi!=N-1){
			uE=dev_U[i+1+N*j];
			}

			//handle x right boundary points
			if(gi==N-1){
			uE=uC;
			pC=-dev_P[i-1+(N-1)*j]; 
			}


			//Handle the up interior point
			if(gj!=0 ){
				uN=dev_U[i+N*(j-1)];
			}

			//Handle the y top boundary case
			if(gj==0 ){
				uN=-uC;
			}

			//Handle the y down interior case
			if(gj!=(N-1)-1 ){
				uS=dev_U[i+N_col*(j+1)];
			}

			//Handle the y down boundary case
			if(gj==(N-1)-1 ){
				uS=-uC;
			}

			// Compute Residual 
				resU= mu*(uE + uW - 2*uC)+ mu*(uN + uS - 2*uC) - dx*(pC-pW);
				dev_U[i+N*j]=dev_U[i+N*j]+ omega*resU;

			//Check if this is the biggest residual 


			}

		}

	}


*/


//}



int main(int argc, char* argv[]){

//Since the error tolerance is 1e-5 I will just use floats instead of doubles to reduce memory transfer times
// and number of blocks

// Get Passed in Params
int	   N=atoi(argv[1]); 
double mu=atof(argv[2]); 
double omega=atof(argv[3]); 
double P_const=atof(argv[4]); 
double tol=atof(argv[5]); 
double dx=1/N ;


//as in the book so in my code
	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.multiProcessorCount =13; 
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(hipSetDevice(dev));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,dev));
// Now props has the info I need to// my code

//Host Data


	double* u=(double*) malloc((N-1)*N*sizeof(double)); 				
	double* v=(double*) malloc((N-1)*N*sizeof(double)); 	
	double* p=(double*) malloc((N-1)*(N-1)*sizeof(double)); 		
/*
doesnt work?
	memset(u , 0, (N-1)*N*sizeof(double));	
	memset(v , 0, (N-1)*N*sizeof(double));	
	memset(p , 0, (N-1)*(N-1)*sizeof(double));	

*/
	for(int i=0; i<(N*(N-1)); i++ ){
		u[i]=0;
		v[i]=0;
		if(i<(N-1)*(N-1))
		p[i]=0;	
	}





	double currentResidual=100;

//Compute the number of blocks I need.
	int storageOfRow=N*sizeof(double);
	int totalFastMemory=prop.sharedMemPerBlock;
	int rowsPerBlock=(total_fast_memory/storageOfRow);

	//So we want each block to carry some of U V and P and they need additional storage
	// for the rows bove and below so u has N rows but we assume that rowsperblock is 2 less than it actually is 
	//then we have N for U and (N-1) for v and P 


//Handle Device Data 
	
	double* dev_u;	
	double* dev_v; 	
	double* dev_p; 

	double*	dev_N;
	double* dev_mu; 
	double* dev_dx; 
	double* dev_omega; 
	double* dev_P_const; 
	double* dev_tol; 

	double* dev_maxResU; 
	double* dev_maxResV; 
	double* dev_maxResP; 

	int numberOfBlocksU=N/(rowsPerBlock-2) ; 
	int numberOfBlocksV=(N-1)/(rowsPerBlock-2) ; 
	int numberOfBlocksP=(N-1)/(rowsPerBlock-2) ; 

	int* dev_numberOfBlocksU; 
	int* dev_numberOfBlocksV; 
	int* dev_numberOfBlocksP; 



	HANDLE_ERROR(hipMalloc((double*) &dev_u, (N-1)*N,sizeof(double)));
	HANDLE_ERROR(hipMalloc((double*) &dev_v, (N-1)*N,sizeof(double)));
	HANDLE_ERROR(hipMalloc((double*) &dev_p, (N-1)*(N-1),sizeof(double)));


/*
	HANDLE_ERROR(hipMalloc((void*) &dev_N, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void*) &dev_mu, sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_dx, sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_omega, sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_P_const, sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_tol, sizeof(double)));

	HANDLE_ERROR(hipMalloc((void*) &dev_ResU, (N-1)*N*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_ResV, (N-1)*N*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void*) &dev_ResP,(N-1)*(N-1)*sizeof(double)));

	HANDLE_ERROR(hipMalloc((void*) &dev_numberOfBlocksU, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void*) &dev_numberOfBlocksV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void*) &dev_numberOfBlocksP, sizeof(int)));


	HANDLE_ERROR(hipMemcpy( dev_N, &N, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_mu, &mu, sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_dx, &dx, sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_omega, &omega, sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_P_const, &P_const, sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_tol, &tol, sizeof(double), hipMemcpyHostToDevice));

	//hipMemcpy( dev_ResU, &currentResidual, (N-1)*N*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy( dev_ResV, &currentResidual, (N-1)*N*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy( dev_ResP, &currentResidual, (N-1)*(N-1)*sizeof(double), hipMemcpyHostToDevice);

	HANDLE_ERROR(hipMemcpy( dev_numberOfBlocksU, &numberOfBlocksU, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_numberOfBlocksV, &numberOfBlocksV, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( dev_numberOfBlocksP, &numberOfBlocksP, sizeof(int), hipMemcpyHostToDevice));




*/


	free(u);
	free(v);
	free(p);

/*
	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_p);




	hipFree(dev_numberOfBlocksU);
	hipFree(dev_numberOfBlocksV);
	hipFree(dev_numberOfBlocksP);

	hipFree(dev_N);
	hipFree(dev_mu);
	hipFree(dev_dx);
	hipFree(dev_omega);
	hipFree(dev_P_const);
	hipFree(dev_omega);
	hipFree(dev_tol);
	hipFree(dev_maxResU);
	hipFree(dev_maxResV);
	hipFree(dev_maxResP);
*/

}